#include<stdio.h>
#include<hip/hip_runtime.h>

#define N 3

__global__ void square_matrix(int *A, int *result) {

	unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned i = index / N;
	unsigned j = index % N;
	
	for(unsigned kk = 0; kk < N; kk++) 
		result[index] += A[i*N + kk]*A[kk*N + j];
	
}

int main() {
	int Arr[N*N], result[N*N];
	int *dArr, *dresult;
	
	for(int ii = 0; ii < N*N; ii++)
		Arr[ii] = ii;
		
	hipMalloc(&dArr, N*N*sizeof(int));
	hipMemcpy(dArr, Arr, N*N*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc(&dresult, N*N*sizeof(int));
	
	square_matrix<<<1, 9>>>(dArr, dresult);
	hipDeviceSynchronize();
	
	hipMemcpy(result, dresult, N*N*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int jj = 0; jj < N*N; jj++) printf("%d ", result[jj]);
	
	return 0;
	
}
	
